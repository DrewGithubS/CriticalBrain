#include "hip/hip_runtime.h"
#include <cstdint>

#include "NeuralNetCUDA.h"

const uint32_t THREADSPERBLOCK = 1024;	
#define BlockCount(x) ((x + THREADSPERBLOCK - 1)/THREADSPERBLOCK)


__global__ void d_setupRand(hiprandState *state,
						    int neurons) {

	int index = threadIdx.x + blockDim.x * blockIdx.x;
	
	if(index < neurons) {
		hiprand_init(1234, index, 0, &state[index]);
	}
}

__global__ void d_randomizeNeurons(hiprandState * curandStates,
								   float * activationThresholds,
								   int16_t * partitionLocs,
								   float minValue,
								   float maxValue,
								   int partitions,
								   int neurons){

	int index = threadIdx.x + blockDim.x*blockIdx.x;

	if(index < neurons) {
		activationThresholds[index] = hiprand_uniform(curandStates + index);
		activationThresholds[index] *= (maxValue - minValue + 0.999999);
		activationThresholds[index] += minValue;
		partitionLocs[index] = index/partitions;
	}
}

/*
          zyx
0  1  2   000 001 002
3  4  5   010 011 012
6  7  8   020 021 022

9  10 11  100 101 102
12 13 14  110 111 112
15 16 17  120 121 122

18 19 20  200 201 202
21 22 23  210 211 212
24 25 26  220 221 222

x = (n % 3)
y = ((n / 3) % 3)
z = (n / (3 * 3))
*/

__global__ void d_createRandomConnections(hiprandState * curandStates,
										  float minWeight,
										  float maxWeight,
										  int32_t * forwardConnections,
										  int32_t * connectionsWeights,
								   		  int partitions,
								   		  int neuronsPerPartition,
								   		  int neuronCount,
								   		  int connectionsPerNeuron,
								   		  int connectionCount) {

	int index = threadIdx.x + blockDim.x*blockIdx.x;

	if(index < connectionCount) {
		int neuronIndex = index / connectionsPerNeuron;
		int partitionIndex = neuronIndex / neuronsPerPartition;
		int partitionX = (partitionIndex % partitions); 
		int partitionY = (partitionIndex / partitions) % partitions;
		int partitionZ = (partitionIndex / (partitions * partitions));
		
		float minValueX = partitionX == 0 ? 0 : -1;
		float minValueY = partitionY == 0 ? 0 : -1;
		float minValueZ = partitionZ == 0 ? 0 : -1;

		float maxValueX = partitionX == (partitions - 1) ? 0 : 1;
		float maxValueY = partitionY == (partitions - 1) ? 0 : 1;
		float maxValueZ = partitionZ == (partitions - 1) ? 0 : 1;

		

		float x_f = hiprand_uniform(curandStates + index);
	    x_f *= (maxValueX - minValueX + 0.999999);
	    x_f += minValueX;
	    int dx = (int) truncf(x_f);

	    float y_f = hiprand_uniform(curandStates + index);
	    y_f *= (maxValueY - minValueY + 0.999999);
	    y_f += maxValueY;
	    int dy = (int) truncf(y_f);

	    float z_f = hiprand_uniform(curandStates + index);
	    z_f *= (maxValueZ - minValueZ + 0.999999);
	    z_f += maxValueZ;
	    int dz = (int) truncf(z_f);

	    int neuronPartitionIndex = 
	    		(partitionZ + dz) * partitions * partitions +
				(partitionY + dy) * partitions + 
				(partitionX + dx);

        neuronPartitionIndex *= neuronsPerPartition;

		int newNeuronIndex;
		do {
			float z_f = hiprand_uniform(curandStates + index);
		    z_f *= (neuronsPerPartition - 0 + 0.999999);
		    z_f += 0;
		    newNeuronIndex = neuronPartitionIndex + (int) truncf(z_f);
		} while(newNeuronIndex == neuronIndex);

		forwardConnections[index] = neuronIndex;

		float weight = hiprand_uniform(curandStates + index);
	    weight *= (maxWeight - minWeight + 0.999999);
	    weight += minWeight;
		
	    connectionsWeights[index] = weight;
	}
}

void randomizeNeurons(hiprandState * curandStates,
					  float * activationThresholds,
					  int16_t * partitionLocs,
					  float minValue,
					  float maxValue,
					  int16_t partitions,
					  int neuronsPerPartition) {

	int neurons = partitions * neuronsPerPartition;

	d_setupRand <<< BlockCount(neurons), THREADSPERBLOCK >>> (
		curandStates,
		neurons);

	d_randomizeNeurons <<< BlockCount(neurons), THREADSPERBLOCK >>> (
		curandStates,
		activationThresholds,
		partitionLocs,
		minValue,
		maxValue,
		partitions,
		neurons);
}

void createRandomConnections(hiprandState * curandStates,
							 float minWeight,
							 float maxWeight,
							 int32_t * forwardConnections,
							 int32_t * connectionsWeights,
							 int partitions,
							 int neuronsPerPartition,
							 int neuronCount,
							 int connectionsPerNeuron,
							 int connectionCount) {

	int neurons = partitionCount * neuronsPerPartition;
	int connections = neurons * maxConnectionsPerNeuron;

	d_createRandomConnections <<< 
								  BlockCount(connections),
								  THREADSPERBLOCK >>> (
		curandStates,
		minWeight,
		maxWeight,
		forwardConnections,
		connectionsWeights,
		partitions,
		neuronsPerPartition,
		neuronCount,
		connectionsPerNeuron,
		connectionCount);
}