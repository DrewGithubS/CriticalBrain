#include <iostream>

#include "GPUFunctions.h"

void * gpuMemAlloc(uint32_t bytes) {
	void * output;
	hipError_t err = hipMalloc(&output, bytes);
	if ( err != hipSuccess ) {
		std::cout << hipGetErrorString(err) << std::endl;
		return NULL;
	}

	return output;
};

void memcpyCPUtoGPU(void * to, void * from, size_t size) {
	hipMemcpy(to, from, size, hipMemcpyHostToDevice);
}

void memcpyGPUtoCPU(void * to, void * from, size_t size) {
	hipMemcpy(to, from, size, hipMemcpyDeviceToHost);
}